#include <iostream> 
#include <vector> 
#include <hip/hip_runtime.h>


 
#pragma comment(lib, "cudart") 

 
using std::cerr; 
using std::cout; 
using std::endl; 
using std::exception; 
using std::vector; 
 
static const int MaxSize = 96; 
 
// CUDA kernel: cubes each array value 
__global__ void cubeKernel(float* result, float* data) 
{ 
    int idx = threadIdx.x; 
    float f = data[idx]; 
    result[idx] = f * f * f; 
} 
 
// Initializes data on the host 
void InitializeData(vector<float>& data) 
{ 
    for (int i = 0; i < MaxSize; ++i) 
    { 
        data[i] = static_cast<float>(i); 
    } 
} 
 
// Executes CUDA kernel 
extern "C" void RunCubeKernel(vector<float>& data, vector<float>& result) 
{ 
    const size_t size = MaxSize * sizeof(float); 
 
    // TODO: test for error 
    float* d; 
    float* r; 
    hipError_t hr; 
 
    hr = hipMalloc(reinterpret_cast<void**>(&d), size);            // Could return 46 if device is unavailable. 
    if (hr == cudaErrorDevicesUnavailable) 
    { 
        cerr << "Close all browsers and rerun" << endl; 
        throw std::runtime_error("Close all browsers and rerun"); 
    } 
 
    hr = hipMalloc(reinterpret_cast<void**>(&r), size); 
    if (hr == cudaErrorDevicesUnavailable) 
    { 
        cerr << "Close all browsers and rerun" << endl; 
        throw std::runtime_error("Close all browsers and rerun"); 
    } 
 
    // Copy data to the device 
    hipMemcpy(d, &data[0], size, hipMemcpyHostToDevice); 
 
    // Launch kernel: 1 block, 96 threads 
    // Important: Do not exceed number of threads returned by the device query, 1024 on my computer. 
    cubeKernel<<<1, MaxSize>>>(r, d); 
 
    // Copy back to the host 
    hipMemcpy(&result[0], r, size, hipMemcpyDeviceToHost); 
 
    // Free device memory 
    hipFree(d); 
    hipFree(r); 
} 
 
// Main entry into the program 
/*
int main(void) 
{ 
    cout << "In main." << endl; 
 
    // Create sample data 
    vector<float> data(MaxSize); 
    InitializeData(data); 
 
    // Compute cube on the device 
    vector<float> cube(MaxSize); 
    RunCubeKernel(data, cube); 
 
    // Print out results 
    cout << "Cube kernel results." << endl << endl; 
 
    for (int i = 0; i < MaxSize; ++i) 
    { 
        cout << cube[i] << endl; 
    } 
 
    return 0; 
} 
*/