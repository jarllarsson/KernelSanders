#include "hip/hip_runtime.h"
#ifndef RAYTRACEKERNELENTRY_CU
#define RAYTRACEKERNELENTRY_CU


#include <vector> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "RaytraceConstantBuffer.h"
#include "KernelHelper.h"

// device specific
#include "Raytracer.h"
 
#pragma comment(lib, "cudart") 

 
using std::vector; 
 



__constant__ RaytraceConstantBuffer cb[1];



texture<float, 2, hipReadModeElementType> texRef;

__global__ void RaytraceKernel(unsigned char *p_outSurface, 
							   const int p_width, const int p_height, const size_t p_pitch)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    float *pixel;

    // discard pixel if outside
    if (x >= p_width || y >= p_height) return;

    // get a pointer to the pixel at (x,y)
    pixel = (float *)(p_outSurface + y*p_pitch) + 4*x;

	Raytrace(pixel,x,y, p_width, p_height);
}
 
// Executes CUDA kernel 
extern "C" void RunRaytraceKernel(void* p_cb,unsigned char *surface,
			int width, int height, int pitch) 
{ 
	// copy to constant buffer
	hipError_t res = hipMemcpyToSymbol(HIP_SYMBOL(cb), p_cb, sizeof(p_cb));
	KernelHelper::assertAndPrint(res,__FILE__,__FUNCTION__,__LINE__);

	// Set up dimensions
	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

	DEBUGPRINT(( ("\n"+toString(width)+" x "+toString(height)).c_str() ));

    RaytraceKernel<<<Dg,Db>>>((unsigned char *)surface, width, height, pitch);

	res = hipDeviceSynchronize();
	KernelHelper::assertAndPrint(res,__FILE__,__FUNCTION__,__LINE__);

} 

#endif